#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>

// final time of simulation

// FHN cell model parameters
#define A 0.2
#define B 0.2
#define C 3.0
#define I_APP 1.0

// CUDA kernel to simulate FHN cell models
__global__ void fhn_kernel(float* ui, float* vi,float *ki, float* u_solution, float* v_solution, float* t_solution, float DT, int NUM_CELLS, float T_FINAL, int rate, int N) {
    
    
    
    
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // calculate global index of thread
    float u_i = ui[idx];
    float v_i = vi[idx];
    float iapp= ki[idx];
 
    
    
    
    
    if (idx < NUM_CELLS) {
        int step = 0;
        
        float ro=N*rate;
        
        #pragma unroll 
        for (int t_i = 0; t_i < N*rate; t_i++) {
            // update FHN model equations for current cell

            float u_new = u_i + DT * (10*(u_i*(u_i-0.4)*(1-u_i)-v_i + iapp));
            float v_new = v_i + DT * ((u_i*0.04-0.16*v_i));

            // store solution for current time step
            int ind = idx + N * step;

           
           
           
            //if (step == 0 || (t_i) % rate == 0 || (t_i)>=ro) {
            
            
            
            u_solution[ind] = u_i;
            v_solution[ind] = v_i;
            t_solution[ind] = t_i * DT;
            
            
            step= (step == 0 || (t_i) % rate == 0 || (t_i) >= ro)?step+1:step;
            
            
            
            
            //}
            
     
            
                        u_i = u_new;
            v_i = v_new;
        }
    }
}

int main(int argc, char* argv[]) {

    float DT = 0.1;
    float T_FINAL =100;
    int rate = 2;

    if (argc != 4) {
        printf("Usage: program_name arg1 arg2\n");
  
    }
    else {
        DT = atof(argv[2]);
        T_FINAL = atof(argv[1]);
        rate = atoi(argv[3]);
    }
    int N=T_FINAL / (DT * rate);
    //if(T_FINAL / (DT * rate)>int(T_FINAL / (DT * rate)))
      //  N++;
   

    int NUM_CELLS = 0;
    std::ifstream file("u.csv");
    std::string row;
    while (std::getline(file, row)) {
        NUM_CELLS++;
    }

    printf("%d", NUM_CELLS);


    // allocate memory on host for FHN model variables
    float* u_host = (float*)malloc(sizeof(float) * NUM_CELLS);
    float* v_host = (float*)malloc(sizeof(float) * NUM_CELLS);
    float* k_host = (float*)malloc(sizeof(float) * NUM_CELLS);


    float* u_solution = (float*)malloc(sizeof(float) * NUM_CELLS * N);
    float* v_solution = (float*)malloc(sizeof(float) * NUM_CELLS * N);
    float* t_solution = (float*)malloc(sizeof(float) * NUM_CELLS * N);


    std::ifstream file2("u.csv"); // Assuming the CSV contains u,v,k values
    std::string line;
    int i = 0;
    while (std::getline(file2, line) && i < NUM_CELLS) {
        std::stringstream ss(line);
        std::string cell;
        if (std::getline(ss, cell, ',') && i < NUM_CELLS) {
            u_host[i] = std::stof(cell);
        }
        if (std::getline(ss, cell, ',') && i < NUM_CELLS) {
            v_host[i] = std::stof(cell);
        }
        if (std::getline(ss, cell, ',') && i < NUM_CELLS) {
            k_host[i] = std::stof(cell);
        }
        
        i++;
    }
    file2.close();
    
    
    // start measuring time for memory allocation and data transfer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    
    // allocate memory on device for FHN model variables
    float* u_dev, * v_dev, * t_dev;
    hipMalloc((void**)&u_dev, sizeof(float) * NUM_CELLS);
    hipMalloc((void**)&v_dev, sizeof(float) * NUM_CELLS);
    hipMalloc((void**)&t_dev, sizeof(float) * NUM_CELLS);


    float* u_solution_dev, * v_solution_dev, * t_solution_dev;
    hipMalloc((void**)&u_solution_dev, sizeof(float) * NUM_CELLS * N);
    hipMalloc((void**)&v_solution_dev, sizeof(float) * NUM_CELLS * N);
    hipMalloc((void**)&t_solution_dev, sizeof(float) * NUM_CELLS * N);

    // copy FHN model variables from host to device
    hipMemcpy(u_dev, u_host, sizeof(float) * NUM_CELLS, hipMemcpyHostToDevice);
    hipMemcpy(v_dev, v_host, sizeof(float) * NUM_CELLS, hipMemcpyHostToDevice);
    hipMemcpy(t_dev, k_host, sizeof(float) * NUM_CELLS, hipMemcpyHostToDevice);

    // calculate number of CUDA threads and blocks to use
    int threads_per_block = 256;
    int blocks_per_grid = (NUM_CELLS + threads_per_block - 1) / threads_per_block;


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float allocation_time;
    hipEventElapsedTime(&allocation_time, start, stop);

 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // simulate FHN cell models on device using CUDA kernel
    fhn_kernel << <blocks_per_grid, threads_per_block >> > (u_dev, v_dev,t_dev, u_solution_dev, v_solution_dev, t_solution_dev, DT, NUM_CELLS, T_FINAL, rate,N);


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float run_time;
    hipEventElapsedTime(&run_time, start, stop);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy FHN model variables from device to host
    hipMemcpy(u_solution, u_solution_dev, sizeof(float) * NUM_CELLS * N, hipMemcpyDeviceToHost);
    hipMemcpy(v_solution, v_solution_dev, sizeof(float) * NUM_CELLS * N, hipMemcpyDeviceToHost);
    hipMemcpy(t_solution, t_solution_dev, sizeof(float) * NUM_CELLS * N, hipMemcpyDeviceToHost);

    FILE* u_fp, * v_fp, * t_fp,*p_fp;
    u_fp = fopen("outputs/u.csv", "w");
    v_fp = fopen("outputs/v.csv", "w");
    t_fp = fopen("outputs/t.csv", "w");
    p_fp = fopen("outputs/p.csv", "w");

    
    for (int i = 0; i < NUM_CELLS; i++) {
        fprintf(u_fp, "%f", u_host[i]);
        fprintf(v_fp, "%f", v_host[i]);

        for (int j = 1; j < N; j++) {
            int ind = j * NUM_CELLS + i;
            // update FHN model equations for current cell
            float U = u_solution[ind];
            float V = v_solution[ind];
            // print updated values to CSV files
            fprintf(u_fp, ",%f", U);
            fprintf(v_fp, ",%f", V);

        }
        fprintf(u_fp, "\n");
        fprintf(v_fp, "\n");
    }
    fclose(u_fp);
    fclose(v_fp);
    fprintf(t_fp, "%f", t_solution[0]);

    for (int i = 1; i < N; i++) {
        fprintf(t_fp, ", %f ", t_solution[i*NUM_CELLS]);
    }

    fclose(t_fp);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float w_time;
    hipEventElapsedTime(&w_time, start, stop);
    
        
    
    fprintf(p_fp, " %f  ,%f , %f", allocation_time,run_time,w_time);
    fclose(p_fp);
    // free memory
    free(u_host);
    free(v_host);
    free(k_host);
    hipFree(u_dev);
    hipFree(v_dev);
    hipFree(t_dev);

    printf("Run time in seconds: %f\n", run_time/1000);
    return 0;
}
